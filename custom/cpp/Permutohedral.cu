#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include "Permutohedral.hpp"
#include "tensorflow/core/framework/op_kernel.h"
using namespace tensorflow;

#define CHECK_ERROR(error, line) {\
    hipError_t err = error;\
    if(err != hipSuccess){\
        fprintf(stderr, "error %d: %s\n", line, hipGetErrorString(err));\
        exit(1);\
    }\
    }

template<typename T>
__global__ void splatting(
    const T *unary, int d,
    T *values, int np, const int *offset, const T *barycentric, int kernel_d
)
{
    int p = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(; p < np; p += stride)
    {
        for(int j = 0; j < kernel_d+1; j++)
        {
            int o = offset[p * (kernel_d+1) + j] + 1;
            T w = barycentric[p * (kernel_d+1) + j];
            for(int k = 0; k < d; k++)
            {
                values[o * d + k] += w * unary[p * d + k];
            }
        }
    }
}

template<typename T>
__global__ void bluring(
    T *values, T *newval, int d, int kernel_d, int *neighbours, int M, bool reverse
)
{
    int i0 = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if(!reverse)
    {
        for(int j = 0; j < kernel_d+1; j++)
        {
            for(int i = i0; i < M; i += stride)
            {
                T *oldv = &values[(i+1) * d];
                T *newv = &newval[(i+1) * d];
                int *n = &neighbours[(j * M + i) * 2];
                int n1 = n[0] + 1;
                int n2 = n[1] + 1;
                T *n1v = &values[n1 * d];
                T *n2v = &values[n2 * d];
                for(int k = 0; k < d; k++)
                    newv[k] = oldv[k] + 0.5 * (n1v[k] + n2v[k]);
            }
            __syncthreads();
            T *t = newval;
            newval = values;
            values = t;
        }
    }
    else
    {
        for(int j = kernel_d; j >= 0; j--)
        {
            for(int i = i0; i < M; i += stride)
            {
                T *oldv = &values[(i+1) * d];
                T *newv = &newval[(i+1) * d];
                int *n = &neighbours[(j * M + i) * 2];
                int n1 = n[0] + 1;
                int n2 = n[1] + 1;
                T *n1v = &values[n1 * d];
                T *n2v = &values[n2 * d];
                for(int k = 0; k < d; k++)
                    newv[k] = oldv[k] + 0.5 * (n1v[k] + n2v[k]);
            }
            __syncthreads();
            T *t = newval;
            newval = values;
            values = t;
        }
    }
}

template<typename T>
__global__ void slicing(
    T *output, int d, bool add, T weight,
    const T *values, int np, const int *offset, const T *barycentric, int kernel_d
)
{
    T alpha = 1.0 / (1 + pow(2, -kernel_d));
    int p = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(; p < np; p += stride)
    {
        if(!add)
        {
            for(int k = 0; k < d; k++)
                output[p * d + k] = 0;
        }
        for(int j = 0; j < kernel_d+1; j++)
        {
            int o = offset[p * (kernel_d+1) + j] + 1;
            T w = barycentric[p * (kernel_d+1) + j];
            for(int k = 0; k < d; k++)
                output[p * d + k] += weight * w * values[o * d + k] * alpha;
        }
    }
}

template<typename T>
__host__ void Permutohedral<T>::compute(Tensor &output_tensor, const Tensor& unary_tensor, bool add, T weight, bool reverse, const GPUDevice& device)
{
    int batch_size = unary_tensor.dim_size(0),
        height = unary_tensor.dim_size(1),
        width = unary_tensor.dim_size(2),
        d = unary_tensor.dim_size(3);
    int np = height * width;
    const T *unary = unary_tensor.flat<T>().data();
    T *output = output_tensor.flat<T>().data();

    int *offset_kernel;
    T *barycentric_kernel;

    hipMalloc(&offset_kernel, np * (kernel_d+1) * sizeof(int));
    hipMalloc(&barycentric_kernel, np * (kernel_d+1) * sizeof(T));

    for(int b = 0; b < batch_size; b++)
    {
        int kb = b;
        if(kernel_batch == 1)
            kb = 0;
        int M = neighbours[kb].size() / (kernel_d+1);

        T *values_kernel, *newval_kernel;
        int *neighbours_kernel;
        const T *unary_kernel = &unary[b * np * d];
        T *output_kernel = &output[b * np * d];

        hipMalloc(&values_kernel, (M+2) * d * sizeof(T));
        hipMalloc(&newval_kernel, (M+2) * d * sizeof(T));
        hipMalloc(&neighbours_kernel, M * (kernel_d+1) * sizeof(int) * 2);

        hipMemset(values_kernel, 0, (M+2) * d * sizeof(T));
        hipMemcpy(neighbours_kernel, neighbours[kb].data(), M * (kernel_d+1) * sizeof(int) * 2, hipMemcpyHostToDevice);
        hipMemcpy(offset_kernel, &offset_[kb * np * (kernel_d+1)], np * (kernel_d+1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(barycentric_kernel, &barycentric_[kb * np * (kernel_d+1)], np * (kernel_d+1) * sizeof(T), hipMemcpyHostToDevice);
        CHECK_ERROR(hipGetLastError(), __LINE__);

        const int block_count = 512, thread_per_block = 32;
        splatting<<<block_count, thread_per_block>>>(
            unary_kernel, d, values_kernel, np,
            offset_kernel, barycentric_kernel, kernel_d);
        CHECK_ERROR(hipGetLastError(), __LINE__);

        bluring<<<block_count, thread_per_block>>>(
            values_kernel, newval_kernel, d, kernel_d, neighbours_kernel, M, reverse);
        if(kernel_d % 2 == 0)
            std::swap(values_kernel, newval_kernel);
        CHECK_ERROR(hipGetLastError(), __LINE__);
        
        slicing<<<block_count, thread_per_block>>>(
            output_kernel, d, add, weight, values_kernel, np,
            offset_kernel, barycentric_kernel, kernel_d);
        CHECK_ERROR(hipGetLastError(), __LINE__);

        hipFree(values_kernel);
        hipFree(newval_kernel);
        hipFree(neighbours_kernel);
    }
    hipFree(offset_kernel);
    hipFree(barycentric_kernel);
}

template class Permutohedral<float>;
template class Permutohedral<double>;

