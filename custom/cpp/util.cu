#include "tensorflow/core/framework/tensor.h"
using namespace tensorflow;

template<class T>
__host__ void get_tensor_gpu(T *&output, const T* d, int size)
{
    output = new T[size];
    hipMemcpy(output, d, size * sizeof(T), hipMemcpyDeviceToHost);
}

template void get_tensor_gpu(float *&, const float*, int);
template void get_tensor_gpu(double *&, const double*, int);

